
#include <hip/hip_runtime.h>
#include <stdio.h>

// 为什么传入指针？
__global__ void add(int *a, int *b, int *c, int num)
{
    int i = threadIdx.x;
    if (i < num)
    {
        c[i] = a[i] + b[i];
        printf("%d + %d == %d\n", a[i], b[i], c[i]);
    }
}

int main(void)
{
    // init data
    int num = 10;
    int a[num], b[num], c[num];
    int *a_gpu, *b_gpu, *c_gpu;

    for (int i = 0; i < num; i++)
    {
        a[i] = i;
        b[i] = i * i;
    }

    // malloc memory  为什么传双重指针？
    hipMalloc((void **)&a_gpu, num * sizeof(int));
    hipMalloc((void **)&b_gpu, num * sizeof(int));
    hipMalloc((void **)&c_gpu, num * sizeof(int));

    // copy data
    hipMemcpy(a_gpu, a, num * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_gpu, b, num * sizeof(int), hipMemcpyHostToDevice);

    // do
    add<<<1, num>>>(a_gpu, b_gpu, c_gpu, num);

    // get data
    hipMemcpy(c, c_gpu, num * sizeof(int), hipMemcpyDeviceToHost);

    // visualization
    for (int i = 0; i < num; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    return 0;
}