
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void P1Malloc(int *p)
{
    printf("p1malloc: %d\n", &p);
    p = (int *)malloc(10);
    printf("p1malloc: %p\n", p);
    printf("p1malloc: %d\n", &p);
    printf("p1malloc: %d\n", sizeof(p));
}

void P2Malloc(void** p)
{
    // *p = malloc(10);
    printf("p2malloc: %p\n", *p);
    printf("p2malloc: %p\n", p);
}

int main()
{
    int Num=10;
    int* a = &Num;
    printf("initial a: %d\n", *a);
    printf("initial1111 a: %p\n", a);
    printf("initial a: %d\n", sizeof(a));
    P1Malloc(a);
    printf("using * a222222: %p\n", a);
    P2Malloc((void**)&a);
    printf("using ** a: %p\n", a);
    printf("using ** a: %p\n", &a);
    printf("using ** a: %d\n", &a);

    return 0;
}
